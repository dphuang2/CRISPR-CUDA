#include "hip/hip_runtime.h"
#include "common/fmt.hpp"
#include "common/utils.hpp"
#include <string>
#include <iostream>
#include <set>
#include <assert.h>
#include <fstream>

using namespace std;

/*
 *these files should exist from the docker image and uploaded build folder
 */
#define GENOME_FILE_PATH "/data/hg38.twobit"
#define GUIDES_FILE_PATH "./src/sequences.twobit"
/*
 *akin to string.format in Python
 */
#define PRINT(...) LOG(info, string(fmt::format(__VA_ARGS__)))
/*
 *20 nucleotides (typical guide length) / 4 nucleotides per byte = 5
 */
#define GUIDE_SIZE_NUCLEOTIDES 20
#define NUCLEOTIDES_PER_BYTE 4
#define GUIDE_SIZE (GUIDE_SIZE_NUCLEOTIDES / NUCLEOTIDES_PER_BYTE)
/*
 *Add another byte to account for the null byte that is appended during
 *ifstream.getline
 */
#define GUIDE_BUFFER_SIZE (GUIDE_SIZE + 1)
/*
 *This is the common edit distance standard for the sgRNA guide
 */
#define EDIT_DISTANCE_THRESHOLD 4
/*
 * The mapping of bits to A, C, T, or G based on the to_2bit.py
 * script in the scripts folder.
 *
 * The mapping is:
 *     T: 0
 *     C: 1
 *     A: 2
 *     G: 3
 */
#define T_IN_BITS 0b00
#define C_IN_BITS 0b01
#define A_IN_BITS 0b10
#define G_IN_BITS 0b11
/*
 *Since we don't want to test naive on entire genome cause its takes too long,
 *we will limit our testing region to this constant
 */
#define GENOME_TEST_LENGTH (1000000 * 10)
/*
 *This is used for predicting the maximum number of matches we might possibly
 *have
 */
#define MATCHES_PER_GUIDE 2000

#define TILE_WIDTH 1024

#define CUDA_CHECK(stmt) checkCuda(stmt, __FILE__, __LINE__);
void checkCuda(hipError_t result, const char *file, const int line) {
    if (result != hipSuccess) {
        PRINT("{}@{}: CUDA Runtime Error: {}", file, line, hipGetErrorString(result));
        exit(-1);
    }
}

typedef set<tuple<int, uint64_t> > results_t;

typedef struct four_nt {
    /*
     * Since we can represent four nucleotides (nt) (A, C, T, or G) as 2 bit
     * values (0b00, 0b01, 0b10, 0b11), we can use one byte to represent 4 nt.
     * This struct encapsulates four nt in a single byte and has some helper
     * functions for comparisons.
     */
    unsigned char one: 2;
    unsigned char two: 2;
    unsigned char three: 2;
    unsigned char four: 2;

    int hamming_distance(four_nt other) {
        return !(one == other.one) +
            !(two == other.two) +
            !(three == other.three) +
            !(four == other.four);
    }

    char bits_to_char(unsigned char bits) {
        switch(bits) {
            case T_IN_BITS:
                return 'T';
            case C_IN_BITS:
                return 'C';
            case A_IN_BITS:
                return 'A';
            case G_IN_BITS:
                return 'G';
        }
        return -1;
    }

    char to_char(int position) {
        assert(position >= 0 && position <= 3);
        switch(position) {
            case 0:
                return bits_to_char(one);
            case 1:
                return bits_to_char(two);
            case 2:
                return bits_to_char(three);
            case 3:
                return bits_to_char(four);
        }
        return -1;
    }

} four_nt;

/***************************************************************
  NAIVE CPU IMPLEMENTATION
    
Pseudocode for the naive algorithm:

for genome_index in xrange(len(genome) - 19):
    for guide in guides:
        mismatches = 0
        for base_index in xrange(20):
             if genome[genome_index + base_index] != guide[base_index]:
                 mismatches += 1
        if mismatches <= 4:
            results.insert((guide, genome_index))
***************************************************************/
results_t naive_cpu_guide_matching(four_nt * genome, uint64_t genome_length, four_nt * guides, int num_guides) {
    results_t results;
    
    for (uint64_t i = 0; i <= genome_length - GUIDE_SIZE;
            i++) {
        for (int j = 0; j < num_guides; j++) {
            four_nt * guide = &guides[j * GUIDE_BUFFER_SIZE];
            four_nt * genome_base = &genome[i];

            int mismatches = 0;
            for (int k = 0; k < GUIDE_SIZE; k++)
                mismatches += genome_base[k].hamming_distance(guide[k]);

            if (mismatches <= EDIT_DISTANCE_THRESHOLD)
                results.insert(make_tuple(j, i));
        }
    }
    return results;
}

/***************************************************************
  NAIVE GPU IMPLEMENTATION
***************************************************************/

__device__ int two_bit_hamming_distance(char one, char two) {
    /*
     * Take the hamming distance between two bytes for every two bits
     */
    bool a = (one >> 6)          == (two >> 6);
    bool b = ((one >> 4) & 0b11) == ((two >> 4) & 0b11);
    bool c = ((one >> 2) & 0b11) == ((two >> 2) & 0b11);
    bool d = (one & 0b11)        == (two & 0b11);
    return !a + !b + !c + !d;
}


__global__ void naive_gpu_guide_matching_kernel(
        char * genome,
        uint64_t genome_length,
        char * guides,
        int num_guides,
        uint64_t * results,
        int * numResults,
        uint64_t sizeOfResults)
{
    uint64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    int numResultsLocal;
    int resultsIdx;
    char * guide;
    char * genome_base;

    for (uint64_t i = tid; i < genome_length - GUIDE_SIZE; i += gridDim.x * blockDim.x) {
        for (int j = 0; j < num_guides; j++) {
            guide = &guides[j * GUIDE_BUFFER_SIZE];
            genome_base = &genome[i];

            int mismatches = 0;
            for (int k = 0; k < GUIDE_SIZE; k++) {
                mismatches += two_bit_hamming_distance(genome_base[k],
                        guide[k]);
            }

            if (mismatches <= EDIT_DISTANCE_THRESHOLD) {
                numResultsLocal = atomicAdd(numResults, 1);
                if (numResultsLocal < sizeOfResults) {
                    resultsIdx = numResultsLocal * 2;
                    results[resultsIdx] = (uint64_t) j;
                    results[resultsIdx + 1] = i;
                }
            }
        }
    }
}

void naive_gpu_guide_matching(
        char * genome,
        uint64_t genome_length,
        char * guides,
        int num_guides,
        uint64_t * hostResults,
        int * hostNumResults,
        uint64_t sizeOfResults)
{
    uint64_t * deviceResults;
    char * deviceGenome;
    char * deviceGuides;
    int * deviceNumResults;

    CUDA_CHECK(hipMalloc((void **) &deviceResults, sizeOfResults));
    CUDA_CHECK(hipMalloc((void **) &deviceGenome, genome_length));
    CUDA_CHECK(hipMalloc((void **) &deviceGuides, num_guides * GUIDE_BUFFER_SIZE));
    CUDA_CHECK(hipMalloc((void **) &deviceNumResults, sizeof(int)));

    CUDA_CHECK(hipMemcpy(deviceGenome, genome, genome_length, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceGuides, guides, num_guides * GUIDE_BUFFER_SIZE, hipMemcpyHostToDevice));

    dim3 dimGrid(ceil(genome_length / double(TILE_WIDTH)));
    dim3 dimBlock(TILE_WIDTH);
    naive_gpu_guide_matching_kernel<<<dimGrid, dimBlock>>>(
            deviceGenome,
            genome_length,
            deviceGuides,
            num_guides,
            deviceResults,
            deviceNumResults,
            sizeOfResults
            );
    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(hostNumResults, deviceNumResults, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hostResults, deviceResults, sizeOfResults, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(deviceResults));
    CUDA_CHECK(hipFree(deviceGenome));
    CUDA_CHECK(hipFree(deviceGuides));
}

/***************************************************************
  GPU-AWARE IMPLEMENTATION
  
  Possible optimizations:
  - Warp-queue like structure for the results to avoid atomic operation
  - Striding for data coalescing (each thread processes a portion of the genome)
  - Shared memory for the guides 
  - Shared memory for the genome
  - Thread coarsening to take advantage of registers
  - Pre-process the guides into hashes, use rolling hash on genome.
***************************************************************/

/***************************************************************
  HELPER FUNCTIONS
***************************************************************/
four_nt * read_genome(string filename, uint64_t * genome_length) {
    /*
     * Used for reading our file into a byte buffer casted as our four_nt struct.
     */

    /*
     * ios::binary means to read file as is
     * ios::ate means start file pointer at the end so we can get file size
     */
    ifstream file(filename, ios::binary | ios::ate);

    if (!file) {
        PRINT("Could not open file: {}", filename);
        PRINT("Error code: {}", strerror(errno));
    }

    streamsize size = file.tellg();
    file.seekg(0, ios::beg);
    *genome_length = size;

    /*
     * I would love to use vector<four_nt> here but if we want to use the read
     * function, we must pass in a buffer of type char *. I thought about
     * trying to modify the .data() pointer of a vector but that is a
     * read-only pointer and so I am left to casting a char* as four_nt *
     */
    char * buffer = new char [size];
    if (file.read(buffer, size)) {
        return (four_nt *) buffer;
    }

    PRINT("Unable to read {}", filename);
    exit(EXIT_FAILURE);
}

four_nt * read_guides(string filename, int * num_guides) {

    // Assert that these are equal just in case four_nt struct gets changed
    assert(sizeof(four_nt) == sizeof(char));
    
    char * guides;
    ifstream file(filename, ios::binary);

    if (!file) {
        PRINT("Could not open file: {}", filename);
        PRINT("Error code: {}", strerror(errno));
        exit(EXIT_FAILURE);
    }

    // Count number of lines
    string line;
    for (*num_guides = 0; getline(file, line); (*num_guides)++);
    // Move file pointer back to beginning
    file.clear(); // https://stackoverflow.com/questions/5343173/returning-to-beginning-of-file-after-getline
    file.seekg(0, file.beg);

    PRINT("Number of guides to process: {}", *num_guides);

    guides = (char *) malloc(*num_guides * GUIDE_BUFFER_SIZE);

    for(int line_number = 0; line_number < *num_guides; line_number++) {
        file.getline(&guides[line_number * GUIDE_BUFFER_SIZE],
                GUIDE_BUFFER_SIZE);
    }

    PRINT("Successfully read all guides");
    return (four_nt *) guides;
}

void print_sequence(four_nt * sequence) {
    for (int i = 0; i < GUIDE_SIZE; i++) {
        four_nt sequence_byte = sequence[i];
        char one = sequence_byte.to_char(0);
        char two = sequence_byte.to_char(1);
        char three = sequence_byte.to_char(2);
        char four = sequence_byte.to_char(3);
        /*
         *We are printing in reverse order because our machine is little-endian
         */
        printf("%c%c%c%c", four, three ,two, one);
    }
    printf("\n");
}

bool assert_results_equal(results_t cpuResults, uint64_t * gpuResults, int gpuNumResults) {
    /*
     *For comparing the gpu results and the cpuResults
     */
    if (gpuNumResults != cpuResults.size())
        return false;
    for (int i = 0; i < gpuNumResults; i++) {
        int gpuResultsIdx = i * 2;
        int guideIdx = (int) gpuResults[gpuResultsIdx];
        uint64_t genomeIdx = gpuResults[gpuResultsIdx + 1];
        if (!cpuResults.count(make_tuple(guideIdx, genomeIdx)))
            return false;
    }
    return true;
}

int main(int argc, char ** argv) {
    /*
     *Read the genome and guides into memory
     */
    int num_guides;
    uint64_t genome_length;
    four_nt * genome = read_genome(GENOME_FILE_PATH, &genome_length);
    four_nt * guides = read_guides(GUIDES_FILE_PATH, &num_guides);

    for (int i = 0; i < num_guides; i++) {
        four_nt * guide = &guides[i * GUIDE_BUFFER_SIZE];
        PRINT("--------guides[{}]--------", i);
        print_sequence(guide);
    }

    PRINT("Genome length: {} bytes", genome_length);

    /*
     *Instantiate our results variables
     */
    results_t results_truth;
    uint64_t sizeOfResults = num_guides * 2 * MATCHES_PER_GUIDE * sizeof(uint64_t);
    uint64_t * hostResults = (uint64_t *) malloc(sizeOfResults);
    int hostNumResults;

    timer_start("Naive CPU");
    results_truth = naive_cpu_guide_matching(genome, GENOME_TEST_LENGTH, guides, num_guides);
    timer_stop();

    PRINT("Size of CPU results: {}", results_truth.size());

    timer_start("Naive GPU");
    naive_gpu_guide_matching(
            (char *) genome,
            GENOME_TEST_LENGTH,
            (char *) guides,
            num_guides,
            hostResults,
            &hostNumResults,
            sizeOfResults);
    timer_stop();

    PRINT("Size of GPU results: {}", hostNumResults);

    timer_start("Comparing GPU results with CPU results");
    bool equal = assert_results_equal(results_truth, hostResults, hostNumResults);
    timer_stop();
    if (equal)
        PRINT("Results were equal, yay!");
    else
        PRINT("Results were not equal, boo :(");

    /*
     *Free up any dynamic memory
     */
    delete[] genome;
    free(guides);
    free(hostResults);
    return EXIT_SUCCESS;
}
